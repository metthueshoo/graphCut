#include "hip/hip_runtime.h"
#include "GraphCuts.h"

int GraphCuts::graphCutsInit(int widthGrid, int heightGrid, int labels)
{
	deviceCount = checkDevice();

	printf("No. of devices %d\n", deviceCount);
	if (deviceCount < 1)
		return -1;

	int cuda_device = 0;

	hipSetDevice(cuda_device);

	hipDeviceProp_t device_properties;

	(hipGetDeviceProperties(&device_properties, cuda_device));

	if ((3 <= device_properties.major) && (device_properties.minor < 1))
		deviceCheck = 2;
	else
	if ((3 <= device_properties.major) && (device_properties.minor >= 1))
		deviceCheck = 1;
	else
		deviceCheck = 0;

	width = widthGrid;
	height = heightGrid;
	num_Labels = labels;

	blocks_x = 1;
	blocks_y = 1;
	num_of_blocks = 1;

	num_of_threads_per_block = 256;
	threads_x = 32;
	threads_y = 8;

	width1 = threads_x * ((int)ceil((float)width / (float)threads_x));
	height1 = threads_y * ((int)ceil((float)height / (float)threads_y));

	graph_size = width * height;
	graph_size1 = width1 * height1;
	size_int = sizeof(int)* graph_size1;

	blocks_x = (int)((ceil)((float)width1 / (float)threads_x));
	blocks_y = (int)((ceil)((float)height1 / (float)threads_y));

	num_of_blocks = (int)((ceil)((float)graph_size1 / (float)num_of_threads_per_block));

	h_mem_init();
	d_mem_init();
	cueValues = 0;

	return deviceCheck;
}


int GraphCuts::checkDevice()
{
	int deviceCount;

	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		return -1;
	}


	return deviceCount;
}

void GraphCuts::h_mem_init()
{
	h_reset_mem = (int*)malloc(sizeof(int)* graph_size1);
	h_graph_height = (int*)malloc(size_int);
	pixelLabel = (int*)malloc(size_int);
	h_pixel_mask = (bool*)malloc(sizeof(bool)* graph_size1);

	for (int i = 0; i < graph_size1; i++)
	{
		pixelLabel[i] = 0;
		h_graph_height[i] = 0;
	}

	for (int i = 0; i < graph_size1; i++)
	{
		h_reset_mem[i] = 0;
	}
}


void GraphCuts::d_mem_init()
{
	hipMalloc((void**)&d_left_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&d_right_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&d_down_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&d_up_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&d_push_reser, sizeof(int)* graph_size1);
	hipMalloc((void**)&d_sink_weight, sizeof(int)* graph_size1);

	hipMalloc((void**)&s_left_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&s_right_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&s_down_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&s_up_weight, sizeof(int)* graph_size1);
	hipMalloc((void**)&s_push_reser, sizeof(int)* graph_size1);
	hipMalloc((void**)&s_sink_weight, sizeof(int)* graph_size1);

	(hipMalloc((void**)&d_stochastic, sizeof(int)* num_of_blocks));
	(hipMalloc((void**)&d_stochastic_pixel, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_terminate, sizeof(int)* num_of_blocks));


	(hipMalloc((void**)&d_pull_left, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_pull_right, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_pull_down, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_pull_up, sizeof(int)* graph_size1));

	(hipMalloc((void**)&d_graph_heightr, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_graph_heightw, sizeof(int)* graph_size1));
	(hipMalloc((void**)&d_relabel_mask, sizeof(int)* graph_size1));

	(hipMalloc((void**)&d_pixel_mask, sizeof(bool)*graph_size1));
	(hipMalloc((void**)&d_over, sizeof(bool)* 1));
	(hipMalloc((void**)&d_counter, sizeof(int)));

	(hipMalloc((void **)&dPixelLabel, sizeof(int)* width1 * height1));
	(hipMemcpy(d_left_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	(hipMemcpy(d_right_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	(hipMemcpy(d_down_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	(hipMemcpy(d_up_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	(hipMemcpy(d_push_reser, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	(hipMemcpy(d_sink_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));

	h_relabel_mask = (int*)malloc(sizeof(int)*width1*height1);

	h_stochastic = (int *)malloc(sizeof(int)* num_of_blocks);
	h_stochastic_pixel = (int *)malloc(sizeof(int)* graph_size1);



	for (int i = 0; i < graph_size1; i++)
		h_relabel_mask[i] = 1;


	(hipMemcpy(d_relabel_mask, h_relabel_mask, sizeof(int)* graph_size1, hipMemcpyHostToDevice));

	int *dpixlab = (int*)malloc(sizeof(int)*width1*height1);

	for (int i = 0; i < width1 * height1; i++)
	{
		dpixlab[i] = 0;
		h_stochastic_pixel[i] = 1;
	}

	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 1;
	}

	(hipMemcpy(d_stochastic, h_stochastic, sizeof(int)* num_of_blocks, hipMemcpyHostToDevice));
	(hipMemcpy(d_terminate, h_stochastic, sizeof(int)* num_of_blocks, hipMemcpyHostToDevice));
	(hipMemcpy(d_stochastic_pixel, h_stochastic_pixel, sizeof(int)* graph_size1, hipMemcpyHostToDevice));


	(hipMemcpy(dPixelLabel, dpixlab, sizeof(int)* width1 * height1, hipMemcpyHostToDevice));

	free(dpixlab);
}

int GraphCuts::graphCutsSetupDataTerm()
{
	if (deviceCheck < 1)
		return -1;

	(hipMalloc((void **)&dDataTerm, sizeof(int)* width * height * num_Labels));

	(hipMemcpy(dDataTerm, dataTerm, sizeof(int)* width * height * num_Labels, hipMemcpyHostToDevice));

	return 0;
}


int GraphCuts::graphCutsSetupSmoothTerm()
{
	if (deviceCheck < 1)
		return -1;

	(hipMalloc((void **)&dSmoothTerm, sizeof(int)* num_Labels * num_Labels));

	(hipMemcpy(dSmoothTerm, smoothTerm, sizeof(int)* num_Labels * num_Labels, hipMemcpyHostToDevice));

	return 0;
}

int GraphCuts::graphCutsSetupHCue()
{

	if (deviceCheck < 1)
		return -1;

	(hipMalloc((void **)&dHcue, sizeof(int)* width * height));

	(hipMemcpy(dHcue, hCue, sizeof(int)* width * height, hipMemcpyHostToDevice));

	cueValues = 1;

	return 0;
}

int GraphCuts::graphCutsSetupVCue()
{
	if (deviceCheck < 1)
		return -1;

	(hipMalloc((void **)&dVcue, sizeof(int)* width * height));

	(hipMemcpy(dVcue, vCue, sizeof(int)* width * height, hipMemcpyHostToDevice));

	return 0;
}

int GraphCuts::graphCutsSetupGraph()
{

	if (deviceCheck < 1)
		return -1;

	int alpha_label = 1;

	for (int i = 0; i < graph_size1; i++)
	{
		h_reset_mem[i] = 0;
		h_graph_height[i] = 0;
	}

	int blockEdge = (int)((ceil)((float)(width * height) / (float)256));
	dim3 block_weight(256, 1, 1);
	dim3 grid_weight(blockEdge, 1, 1);

	if (cueValues == 1)
	{
		CudaWeightCue <<< grid_weight, block_weight >>>(alpha_label, d_left_weight, d_right_weight, d_down_weight,
			d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm,
			dSmoothTerm, dHcue, dVcue, width, height, 2);
	}
	else
	{
		CudaWeight <<< grid_weight, block_weight >>>(alpha_label, d_left_weight, d_right_weight, d_down_weight,
			d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm,
			dSmoothTerm, width, height, 2);

	}

	int *temp_left_weight, *temp_right_weight, *temp_down_weight, *temp_up_weight, *temp_source_weight, *temp_terminal_weight;

	(hipMalloc((void **)&temp_left_weight, sizeof(int)* graph_size1));
	(hipMalloc((void **)&temp_right_weight, sizeof(int)* graph_size1));
	(hipMalloc((void **)&temp_down_weight, sizeof(int)* graph_size1));
	(hipMalloc((void **)&temp_up_weight, sizeof(int)* graph_size1));
	(hipMalloc((void **)&temp_source_weight, sizeof(int)* graph_size1));
	(hipMalloc((void **)&temp_terminal_weight, sizeof(int)* graph_size1));

	int blockEdge1 = (int)((ceil)((float)(width1 * height1) / (float)256));
	dim3 block_weight1(256, 1, 1);
	dim3 grid_weight1(blockEdge1, 1, 1);

	adjustedgeweight <<<grid_weight1, block_weight1 >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser,
		d_sink_weight, temp_left_weight, temp_right_weight, temp_down_weight, temp_up_weight,
		temp_source_weight, temp_terminal_weight, width, height, graph_size, width1,
		height1, graph_size1);

	copyedgeweight <<<grid_weight1, block_weight1 >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight,
		temp_left_weight, temp_right_weight, temp_down_weight, temp_up_weight, temp_source_weight,
		temp_terminal_weight, d_pull_left, d_pull_right, d_pull_down, d_pull_up, d_relabel_mask,
		d_graph_heightr, d_graph_heightw, width, height, graph_size, width1, height1, graph_size1);

	(hipFree(temp_left_weight));
	(hipFree(temp_right_weight));
	(hipFree(temp_up_weight));
	(hipFree(temp_down_weight));
	(hipFree(temp_source_weight));
	(hipFree(temp_terminal_weight));
	return 0;
}

int GraphCuts::graphCutsAtomicOptimize()
{
	if (deviceCheck < 1)
	{
		return -1;
	}

	graphCutsAtomic();

	bfsLabeling();

	return 0;

}

int GraphCuts::graphCutsStochasticOptimize()
{
	if (deviceCheck < 1)
	{
		return -1;
	}

	graphCutsStochastic();

	bfsLabeling();

	return 0;

}

void GraphCuts::graphCutsAtomic()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	bool finish = true;

	counter = num_of_blocks;

	int numThreadsEnd = 256, numBlocksEnd = 1;
	if (numThreadsEnd > counter)
	{
		numBlocksEnd = 1;
		numThreadsEnd = counter;
	}
	else
	{
		numBlocksEnd = (int)ceil(counter / (double)numThreadsEnd);
	}

	dim3 End_block(numThreadsEnd, 1, 1);
	dim3 End_grid(numBlocksEnd, 1, 1);

	int *d_counter;

	bool *d_finish;
	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 0;
	}

	(hipMalloc((void**)&d_counter, sizeof(int)));
	(hipMalloc((void**)&d_finish, sizeof(bool)));

	(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	counter = 0;
	int *d_relabel;

	(hipMalloc((void**)&d_relabel, sizeof(int)));

	int h_relabel = 0;

	int block_num = width1 / 32;

	int *d_block_num;

	(hipMalloc((void**)&d_block_num, sizeof(int)));
	(hipMemcpy(d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));

	int h_count_blocks = num_of_blocks;
	int *d_count_blocks;

	(hipMalloc((void**)&d_count_blocks, sizeof(int)));
	(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));

	h_count_blocks = 0;



	(hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));

	counter = 1;
	kernel_push1_start_atomic <<<grid_push, block_push >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
		d_sink_weight, d_push_reser,
		d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
		graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish);

	int h_terminate_condition = 0;
	(hipDeviceSynchronize());
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	do
	{

		if (counter % 10 == 0)
		{
			finish = true;
			(hipMemcpy(d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1 <<<grid_push, block_push >>>(d_push_reser, s_push_reser, d_count_blocks, d_finish, d_block_num, width1);
			(hipMemcpy(&finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
			if (finish == false)
				h_terminate_condition++;
		}
		if (counter % 11 == 0)
		{
			(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0;
			(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_atomic2 <<<grid_push, block_push >>>(d_terminate, d_push_reser, s_push_reser, d_block_num, width1);

			kernel_End <<<End_grid, End_block >>>(d_terminate, d_count_blocks, d_counter);

		}

		if (counter % 2 == 0)
		{

			kernel_push1_atomic <<<grid_push, block_push >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1);

			kernel_relabel_atomic <<<grid_push, block_push >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1);
		}
		else
		{
			kernel_push1_atomic <<<grid_push, block_push >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1);
			kernel_relabel_atomic <<<grid_push, block_push >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1);

		}
		counter++;
	} while (h_terminate_condition != 2);

	(hipEventRecord(stop, 0));
	(hipEventSynchronize(stop));

}


void GraphCuts::graphCutsStochastic()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	bool finish = true;

	counter = num_of_blocks;

	int numThreadsEnd = 256, numBlocksEnd = 1;
	if (numThreadsEnd > counter)
	{
		numBlocksEnd = 1;
		numThreadsEnd = counter;
	}
	else
	{
		numBlocksEnd = (int)ceil(counter / (double)numThreadsEnd);
	}

	dim3 End_block(numThreadsEnd, 1, 1);
	dim3 End_grid(numBlocksEnd, 1, 1);




	bool *d_finish;
	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 0;
	}

	(hipMalloc((void**)&d_counter, sizeof(int)));
	(hipMalloc((void**)&d_finish, sizeof(bool)));

	(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	counter = 0;
	int *d_relabel;

	(hipMalloc((void**)&d_relabel, sizeof(int)));

	int h_relabel = 0;


	int block_num = width1 / 32;

	int *d_block_num;

	(hipMalloc((void**)&d_block_num, sizeof(int)));
	(hipMemcpy(d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));


	int h_count_blocks = num_of_blocks;
	int *d_count_blocks;

	(hipMalloc((void**)&d_count_blocks, sizeof(int)));
	(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));

	h_count_blocks = 0;

	(hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));

	counter = 1;
	kernel_push1_start_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
		d_sink_weight, d_push_reser,
		d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
		graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish);
	int h_terminate_condition = 0;
	(hipDeviceSynchronize());
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	do
	{
		if (counter % 10 == 0)
		{
			finish = true;
			(hipMemcpy(d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1 << <grid_push, block_push >> >(d_push_reser, s_push_reser, d_count_blocks, d_finish, d_block_num, width1);
			(hipMemcpy(&finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
		}
		if (counter % 11 == 0)
		{
			(hipMemset(d_stochastic, 0, sizeof(int)*num_of_blocks));
			(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0;
			(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_stochastic2 << <grid_push, block_push >> >(d_terminate, d_relabel_mask, d_push_reser, s_push_reser, d_stochastic, d_block_num, width1);

			kernel_End << <End_grid, End_block >> >(d_terminate, d_count_blocks, d_counter);

			if (finish == false && counter % 121 != 0 && counter > 0)
				h_terminate_condition++;

		}
		if (counter % 2 == 0)
		{

			kernel_push1_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

			kernel_relabel_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

		}
		else
		{
			kernel_push1_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

			kernel_relabel_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

		}
		counter++;
	} while (h_terminate_condition == 0 && counter < 500);


	(hipEventRecord(stop, 0));
	(hipEventSynchronize(stop));
	float time;
	(hipEventElapsedTime(&time, start, stop));
	printf("TT Cuts :: %f ms\n", time);

}

void GraphCuts::bfsLabeling()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	(hipMemcpy(d_graph_heightr, h_graph_height, size_int, hipMemcpyHostToDevice));

	for (int i = 0; i < graph_size; i++)
		h_pixel_mask[i] = true;

	(hipMemcpy(d_pixel_mask, h_pixel_mask, sizeof(bool)* graph_size1, hipMemcpyHostToDevice));

	kernel_bfs_t << <d_grid, d_block, 0 >> >(d_push_reser, d_sink_weight, d_graph_heightr, d_pixel_mask, graph_size, width, height, graph_size1, width1, height1);
	counter = 1;

	(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	do
	{
		h_over = false;

		(hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice));

		kernel_bfs << < d_grid, d_block, 0 >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_graph_heightr, d_pixel_mask,
			graph_size, width, height, graph_size1, width1, height1, d_over, d_counter);

		(hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost));

		counter++;

		(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	} while (h_over);

	(hipMemcpy(h_graph_height, d_graph_heightr, size_int, hipMemcpyDeviceToHost));
}


int GraphCuts::graphCutsGetResult()
{
	if (deviceCheck < 1)
		return -1;

	int alpha = 1;

	for (int i = 0; i < graph_size1; i++)
	{
		int row_here = i / width1, col_here = i % width1;
		if (h_graph_height[i]>0 && row_here < height && row_here > 0 && col_here < width && col_here > 0) {
			pixelLabel[i] = alpha;
		}
	}

	return 0;

}

void GraphCuts::graphCutsFreeMem()
{
	free(h_reset_mem);
	free(h_graph_height);
	free(pixelLabel);
	free(h_pixel_mask);

	free(h_relabel_mask);
	free(h_stochastic);
	free(h_stochastic_pixel);

	free(hCue);
	free(vCue);
	free(dataTerm);
	free(smoothTerm);

	(hipFree(d_left_weight));
	(hipFree(d_right_weight));
	(hipFree(d_down_weight));
	(hipFree(d_up_weight));
	(hipFree(d_sink_weight));
	(hipFree(d_push_reser));

	(hipFree(d_pull_left));
	(hipFree(d_pull_right));
	(hipFree(d_pull_down));
	(hipFree(d_pull_up));

	(hipFree(d_graph_heightr));
	(hipFree(d_graph_heightw));

	(hipFree(s_left_weight));
	(hipFree(s_right_weight));
	(hipFree(s_down_weight));
	(hipFree(s_up_weight));
	(hipFree(s_push_reser));
	(hipFree(s_sink_weight));


	(hipFree(d_stochastic));
	(hipFree(d_stochastic_pixel));
	(hipFree(d_terminate));

	(hipFree(d_relabel_mask));

	(hipFree(d_pixel_mask));
	(hipFree(d_over));
	(hipFree(d_counter));

	(hipFree(dPixelLabel));
}
